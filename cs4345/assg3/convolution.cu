#include "hip/hip_runtime.h"
//============================================================
// STUDENT NAME: Eugene
// MATRIC NO.  : A0116631N
// NUS EMAIL   : eugene@comp.nus.edu.sg
// COMMENTS TO GRADER:
// <comments to grader, if any>
//
// ============================================================
//
// FILE: convolution.cu


// Include files from C standard library.
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

// Includes CUDA.
#include <hip/hip_runtime.h>

// Includes helper functions from CUDA Samples SDK.
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples



/////////////////////////////////////////////////////////////////////////////
// CONSTANTS & GLOBAL VARIABLES
/////////////////////////////////////////////////////////////////////////////

// FILTER_WIDTH must be odd, and BLOCK_SIZE >= FILTER_WIDTH.
#define FILTER_WIDTH    249

// Number of CUDA threads per thread block. BLOCK_SIZE >= FILTER_WIDTH.
#define BLOCK_SIZE      256

// Number of elements in the data.
// Note that DATA_SIZE is always a multiple of BLOCK_SIZE.
#define DATA_SIZE       (2048 * BLOCK_SIZE)

// Number of CUDA thread blocks. 
#define NUM_BLOCKS      ( ( (DATA_SIZE) + (BLOCK_SIZE) - 1 ) / (BLOCK_SIZE) )




//===========================================================================
// CUDA Kernel 1. 
// Does not use shared memory.
// Does not care about memory coalesces.
//
// Compute the convolution of the data and the filter.
// Filter width (filterWidth) must be odd, and the filter's
// origin is the center element, that is the element
// filter[ filterWidth/2 ].
//
// For each output element output[i] that does not have enough input 
// data elements in its neighborhood (that is when i < (filterWidth/2) or
// when i >= (dataSize - filterWidth/2)), the output element output[i]
// will have value 0.0.
//
//===========================================================================
__global__ void GPU_Convolve1( const float *data, int dataSize, 
                               const float *filter, int filterWidth,
                               float *output )
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int filterRadius = filterWidth / 2;

    //***********************************************
    //*********** WRITE YOUR CODE HERE **************
    //***********************************************

	// use local memory
    float result = 0.0;

	if (tid >= filterRadius && tid < (dataSize - filterRadius)){
        for ( int offset = 0; offset < filterWidth; offset++ )
            result += filter[offset] * data[ tid - filterRadius + offset ];
	}
	output[tid] = result;
}



//===========================================================================
// CUDA Kernel 2. 
// Use shared memory.
// Care about memory coalesces.
// Care about shared memory conflicts.
//
// Compute the convolution of the data and the filter.
// Filter width (filterWidth) must be odd, and the filter's
// origin is the center element, that is the element
// filter[ filterWidth/2 ].
//
// For each output element output[i] that does not have enough input 
// data elements in its neighborhood (that is when i < (filterWidth/2) or
// when i >= (dataSize - filterWidth/2)), the output element output[i]
// will have value 0.0. 
//
// Assume that filterWidth <= BLOCK_SIZE.
// Assume that dataSize is a multiple of BLOCK_SIZE.
//
//===========================================================================
__global__ void GPU_Convolve2( const float *data, int dataSize, 
                               const float *filter, int filterWidth,
                               float *output )
{
    __shared__ float filterS[ BLOCK_SIZE ];
    __shared__ float dataS[ 3 * BLOCK_SIZE ];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    int filterRadius = filterWidth / 2;

    //***********************************************
    //*********** WRITE YOUR CODE HERE **************
    //***********************************************

	// Load data into Shared Memory (dataS)
	// Shift index to reserve for Halo (edge most) data
    int lindex = tx + filterRadius;

	// { } { } [X] [X] [X] [X] [X] { } { }
    dataS[lindex] = data[tid];

    if (tx < filterRadius){
		// {o} {o} [x] [x] [x] [x] [x] { } { }
        dataS[tx] = data[tid - filterRadius];
		// { } { } [x] [x] [x] [x] [x] {O} {O}
        dataS[lindex + BLOCK_SIZE] = data[tid + BLOCK_SIZE];
    }

	// Load filter into Shared Memory (filterS)
    if (tx < filterWidth)
        filterS[tx] = filter[tx];

	// Ensure All shared memory is loaded
	__syncthreads();

	// use local Memory to store final output
	float result = 0.0;

	if (tid >= filterRadius && tid < (dataSize - filterRadius)){
        for ( int offset = 0; offset < filterWidth; offset++ ){
			// Convolution
			result += filterS[offset] * dataS[ lindex - filterRadius + offset ];
		}
    }

	// transfer local memory into the global memory
	output[tid] = result;
}




//===========================================================================
// CPU version.
//
// Compute the convolution of the data and the filter.
// Filter width (filterWidth) must be odd, and the filter's
// origin is the center element, that is the element
// filter[ filterWidth/2 ].
//
// For each output element output[i] that does not have enough input 
// data elements in its neighborhood (that is when i < (filterWidth/2) or
// when i >= (dataSize - filterWidth/2)), the output element output[i]
// will have value 0.0. 
//
//===========================================================================
static void CPU_Convolve( const float *data, int dataSize, 
                          const float *filter, int filterWidth,
                          float *output )
{
    int filterRadius = filterWidth / 2;

    for ( int i = 0; i < dataSize; i++ ) output[i] = 0.0;

    for ( int i = filterRadius; i < (dataSize - filterRadius); i++ )
        for ( int k = 0; k < filterWidth; k++ )
            output[i] += filter[k] * data[ i - filterRadius + k ];
}



//===========================================================================
// Returns a random value in the range [min, max] from a uniform distribution.
//===========================================================================
inline static double UniformRandom( double min, double max )
{
    return ( ((double)rand()) / RAND_MAX ) * (max - min) + min;
}



//===========================================================================
// Generates a set of random floating-point numbers in the range [min,max] 
// and put them in the the array A.
//===========================================================================
static void GenerateRandomArray( float *A, int numElems, float min, float max )
{
    for ( int i = 0; i < numElems; i++ )
        A[i] = (float) UniformRandom( min, max );
}



//===========================================================================
// Return true iff all corresponding elements in the float arrays A and B 
// are approximately equal (i.e. the absolute difference is within the
// given epsilon).
//===========================================================================
static bool FloatArrayEqual( const float *A, const float *B, int numElems, float epsilon )
{
    for ( int i = 0; i < numElems; i++ )
        if ( fabs( A[i] - B[i] ) > epsilon ) return false;

    return true;
}



void WaitForEnterKeyBeforeExit( void )
{
    fflush( stdin );
    getchar();
}



//===========================================================================
// The main function
//===========================================================================
int main( int argc, char** argv )
{
    atexit( WaitForEnterKeyBeforeExit );

    // Set seed for rand().
    srand( 123 );

    // Use command-line specified CUDA device, otherwise use device with highest Gflops/s.
    int devID = findCudaDevice( argc, (const char **)argv );

    // Create a timer.
    StopWatchInterface *timer = 0;
    sdkCreateTimer( &timer );


//---------------------------------------------------------------------------
// Allocate memory and generate test data.
//---------------------------------------------------------------------------

    // Allocate host memory for filter, input data and result arrays.
    float *h_filter = (float *) malloc( FILTER_WIDTH * sizeof(float) );
    float *h_data = (float *) malloc( DATA_SIZE * sizeof(float) );
    float *h_output = (float *) malloc( DATA_SIZE * sizeof(float) );

    // Allocate host memory for receiving results from the GPU.
    float *d2h_output1 = (float *) malloc( DATA_SIZE * sizeof(float) );
    float *d2h_output2 = (float *) malloc( DATA_SIZE * sizeof(float) );

    // Allocate device memory.
    float *d_filter, *d_data, *d_output;
    checkCudaErrors( hipMalloc( (void**) &d_filter, FILTER_WIDTH * sizeof(float) ) );
    checkCudaErrors( hipMalloc( (void**) &d_data, DATA_SIZE * sizeof(float) ) );
    checkCudaErrors( hipMalloc( (void**) &d_output, DATA_SIZE * sizeof(float) ) );

    // Fill the host filter and data arrays with random floating-point numbers.
    GenerateRandomArray( h_filter, FILTER_WIDTH, 0.0, 1.0 );
    GenerateRandomArray( h_data, DATA_SIZE, 1.0, 5.0 );


//---------------------------------------------------------------------------
// Print some program parameter values.
//---------------------------------------------------------------------------
    printf( "Filter width = %d\n", FILTER_WIDTH );
    printf( "Data size = %d\n", DATA_SIZE );
    printf( "Thread block size = %d\n", BLOCK_SIZE );
    printf( "Number of thread blocks = %d\n", NUM_BLOCKS );
    printf( "\n\n" );


//---------------------------------------------------------------------------
// Perform computation on CPU.
//---------------------------------------------------------------------------
    printf( "CPU COMPUTATION:\n" );

    // Reset and start timer.
    sdkResetTimer( &timer );
    sdkStartTimer( &timer );

    // Compute on CPU.
    CPU_Convolve( h_data, DATA_SIZE, h_filter, FILTER_WIDTH, h_output );

    // Stop timer.
    sdkStopTimer( &timer );
    printf( "Processing time = %.3f ms\n", sdkGetTimerValue( &timer ) );

    // Print some results.
    printf( "First element  = %.8f\n", h_output[0] );
    printf( "Middle element = %.8f\n", h_output[ DATA_SIZE / 2 ] );
    printf( "Last element   = %.8f\n", h_output[ DATA_SIZE - 1 ] );
    printf( "\n\n" );


//---------------------------------------------------------------------------
// Perform computation on GPU using Kernel 1 (not using shared memory).
//---------------------------------------------------------------------------
    printf( "GPU COMPUTATION 1 (not using Shared Memory):\n" );

    // Reset and start timer.
    sdkResetTimer( &timer );
    sdkStartTimer( &timer );

    // Copy host memory to device.
    checkCudaErrors( hipMemcpy( d_filter, h_filter, FILTER_WIDTH * sizeof(float), hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_data, h_data, DATA_SIZE * sizeof(float), hipMemcpyHostToDevice ) );

    // Clear the output array in device memory.
    checkCudaErrors( hipMemset( d_output, 0, DATA_SIZE * sizeof(float) ) );

    // Execute the kernel.
    GPU_Convolve1 <<<NUM_BLOCKS, BLOCK_SIZE>>> ( d_data, DATA_SIZE, d_filter, FILTER_WIDTH, d_output );

    // Check if kernel execution generated any error.
    getLastCudaError( "Kernel execution failed" );

    // Copy result from device memory to host.
    checkCudaErrors( hipMemcpy( d2h_output1, d_output, DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost ) );

    // Stop timer.
    sdkStopTimer( &timer );
    printf( "Processing time = %.3f ms\n", sdkGetTimerValue( &timer ) );

    // Print some results.
    printf( "First element  = %.8f\n", d2h_output1[0] );
    printf( "Middle element = %.8f\n", d2h_output1[ DATA_SIZE / 2 ] );
    printf( "Last element   = %.8f\n", d2h_output1[ DATA_SIZE - 1 ] );

    // Check result with reference result computed by CPU.
    bool equal1 = FloatArrayEqual( h_output, d2h_output1, DATA_SIZE, 0.001f );
    printf( "Verify GPU result... %s\n", (equal1)? "PASS" : "FAIL" );
    printf( "\n\n" );


//---------------------------------------------------------------------------
// Perform computation on GPU using Kernel 2 (using shared memory).
//---------------------------------------------------------------------------
    printf( "GPU COMPUTATION 2 (using Shared Memory):\n" );

    // Reset and start timer.
    sdkResetTimer( &timer );
    sdkStartTimer( &timer );

    // Copy host memory to device.
    checkCudaErrors( hipMemcpy( d_filter, h_filter, FILTER_WIDTH * sizeof(float), hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_data, h_data, DATA_SIZE * sizeof(float), hipMemcpyHostToDevice ) );

    // Clear the output array in device memory.
    checkCudaErrors( hipMemset( d_output, 0, DATA_SIZE * sizeof(float) ) );

    // Execute the kernel.
    GPU_Convolve2 <<<NUM_BLOCKS, BLOCK_SIZE>>> ( d_data, DATA_SIZE, d_filter, FILTER_WIDTH, d_output );

    // Check if kernel execution generated any error.
    getLastCudaError( "Kernel execution failed" );

    // Copy result from device memory to host.
    checkCudaErrors( hipMemcpy( d2h_output2, d_output, DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost ) );

    // Stop timer.
    sdkStopTimer( &timer );
    printf( "Processing time = %.3f ms\n", sdkGetTimerValue( &timer ) );

    // Print some results.
    printf( "First element  = %.8f\n", d2h_output2[0] );
    printf( "Middle element = %.8f\n", d2h_output2[ DATA_SIZE / 2 ] );
    printf( "Last element   = %.8f\n", d2h_output2[ DATA_SIZE - 1 ] );

    // Check result with reference result computed by CPU.
    bool equal2 = FloatArrayEqual( h_output, d2h_output2, DATA_SIZE, 0.001f );
    printf( "Verify GPU result... %s\n", (equal2)? "PASS" : "FAIL" );
    printf( "\n\n" );


//---------------------------------------------------------------------------
// Clean up.
//---------------------------------------------------------------------------
    // Destroy the timer.
    sdkDeleteTimer( &timer );

    // Free up memory.
    free( h_filter );
    free( h_data );
    free( h_output );
    free( d2h_output1 );
    free( d2h_output2 );
    checkCudaErrors( hipFree( d_filter ) );
    checkCudaErrors( hipFree( d_data ) );
    checkCudaErrors( hipFree( d_output ) );

    hipDeviceReset();
}
