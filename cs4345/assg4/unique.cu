#include "hip/hip_runtime.h"
//============================================================
// STUDENT NAME: Eugene
// MATRIC NO.  : A0116631N
// NUS EMAIL   : eugene@comp.nus.edu.sg
// COMMENTS TO GRADER:
// <comments to grader, if any>
//
// ============================================================
//
// FILE: unique.cu


// Include files from C standard library.
#include <stdlib.h>
#include <stdio.h>
#include <string.h>  // For memcpy().
#include <math.h>

// Includes CUDA.
#include <hip/hip_runtime.h>

// Includes helper functions from CUDA Samples SDK.
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples.


// Include files to use Thrust (a C++ template library for CUDA).
// Thrust v1.7.0 is automatically installed with CUDA Toolkit 6.5.
// Read more about Thrust at the GitHub Thrust project page
// (http://thrust.github.com/).

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/scan.h>



/////////////////////////////////////////////////////////////////////////////
// CONSTANTS & GLOBAL VARIABLES
/////////////////////////////////////////////////////////////////////////////

#define NUM_ELEMS       (5*1000000)  // Number of elements in input array.

#define BLOCK_SIZE      256
#define NUM_BLOCKS      ( ( (NUM_ELEMS) + (BLOCK_SIZE) - 1 ) / (BLOCK_SIZE) )

#define ELEM_MIN        1       // Minimum value in input array (must not be negative).
#define ELEM_MAX        100000  // Maximum value in input array (must not be negative).



//===========================================================================
// CUDA kernel used by GPU_Unique().
//
// Given an input sorted integer array, the kernel marks in the output array
// which elements of the input array should be kept/removed, so that if these 
// elements were to be kept/removed, there would be no duplicate elements in 
// the sorted array. We want to remove as few elements as possible from the 
// input array. 
//
// The output of the kernel is an array of 1's and 0's to indicate whether 
// the corresponding elements in the input array should be kept or removed -- 
// a 1 means keep, and 0 means remove. The output array has the same number 
// of elements as the input array.
//
// For example, given the following input array
// 
//        inSortedArray[] = [ 1 1 3 3 3 5 5 7 8 8 ]
//
// the output would be
//
//    outSelectionArray[] = [ 1 0 1 0 0 1 0 1 1 0 ]
//
// so that if we keep only those elements in the input array that have a 1
// in the corresponding location in the output array, we will have the 
// result [ 1 3 5 7 8 ].
// 
// NOTE: You should use shared memory to minimize the number of uncoalesced 
// global memory accesses. Shared memory conflicts must be minimized too.
//===========================================================================
__global__ void Kernel_MarkUnique( int *inSortedArray, int *outSelectionArray,
                                   int numElems )
{
    //***********************************************
    //*********** WRITE YOUR CODE HERE **************
    //***********************************************
	__shared__ int inSortedArrayS[BLOCK_SIZE + 1];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	int lindex = tx + 1;

	inSortedArrayS[lindex] = inSortedArray[tid];
	if (tid > 0 && tx == 0){
		inSortedArrayS[0] = inSortedArray[tid-1];
	}
	__syncthreads();

	if (tid > 0){
		if (inSortedArrayS[lindex] == inSortedArrayS[tx]){
			outSelectionArray[tid] = 0;
		} else {
			outSelectionArray[tid] = 1;
		}
	} else {
		outSelectionArray[tid] = 1;
	}
}



//===========================================================================
// CUDA kernel used by GPU_Unique().
//
// The kernel copies a selected set of elements from the input array to
// specified locations in the output array.
//
// For an input element inArray[i], if selectionArray[i] is 1, then
// the input element is copied to the output array outArray[]. 
// The location in the output array it is copied to is 
// scatterAddressArray[i] + addressOffset.
//
// You can assume that no two elements in the input array inArray[] 
// will be selected and copied to the same location in the output 
// array outArray[].
// 
// NOTE: You do not need to use shared memory, but try to keep the 
// number of uncoalesced global memory accesses to the minimal.
//===========================================================================
__global__ void Kernel_Scatter( int *inArray, int *selectionArray,
                                int *scatterAddressArray, int addressOffset,
                                int *outArray, int numElems )
{
    //***********************************************
    //*********** WRITE YOUR CODE HERE **************
    //***********************************************
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < numElems && selectionArray[tid] == 1){
		outArray[scatterAddressArray[tid] + addressOffset] = inArray[tid];
	}
}



//===========================================================================
// Used by GPU_Unique().
//
// Use Thrust's sort algorithm to sort the input integer array on the GPU,
// in non-decreasing order. The sort is performed in-place, 
// 
// NOTE:
// * The input/output array is already allocated in the device memory.
//===========================================================================
static void GPU_SortIntegerArray( int *d_inoutArray, int numElems )
{
    thrust::device_ptr<int> dev_ptr( d_inoutArray );
    thrust::sort( dev_ptr, dev_ptr + numElems );
}



//===========================================================================
// Used by GPU_Unique().
//
// Use Thrust's scan algorithm to compute the "inclusive" all-prefix sums on the GPU.
// Also produces the sum of all elements in the input array in the output
// parameter *h_outInArraySum.
// 
// NOTE: The input and output arrays are already allocated in the device memory.
//===========================================================================
static void GPU_AllPrefixSums( int *d_inArray, int *d_outArray, int numElems, 
                               int *h_outInArraySum )
{
    thrust::device_ptr<int> in_dev_ptr( d_inArray );
    thrust::device_ptr<int> out_dev_ptr( d_outArray );
    thrust::inclusive_scan( in_dev_ptr, in_dev_ptr + numElems, out_dev_ptr );

    // Get the sum of all the elements in the input array. This can be obtained 
    // from the last element in the all-prefix-sums array.
    checkCudaErrors( hipMemcpy( h_outInArraySum, d_outArray + numElems - 1, 
                                 sizeof(int), hipMemcpyDeviceToHost ) );

    // Using Thrust, the above memory copy can be written as:
    // *h_outInArraySum = out_dev_ptr[ numElems - 1 ];
}



//===========================================================================
// GPU version.
//
// Given an input integer array, the function produces an output array
// which is a sorted version of the input array, but with duplicate 
// elements removed. The output array is sorted in non-decreasing order.
// The function also produces the number of unique elements in the
// output array in the parameter (*numUniqueElems).
//
// For example, if the input array is [ 5 3 7 5 8 3 1 3 1 8 ], the 
// output array would be [ 1 3 5 7 8 ].
//
// When this function is called, sufficient memory storage must have 
// already been allocated for the output array. The safest is to allocate
// as much memory as for the input array.
//
// Here, a scan-and-scatter approach is used to do the stream compaction 
// on the GPU. The following example demonstrates the steps.
//
// (0) Input array:  
//              inputArray[] = [ 5 3 7 5 8 3 1 3 1 8 ]
//
// (1) Sort inputArray[]:   
//             sortedArray[] = [ 1 1 3 3 3 5 5 7 8 8 ]
//
// (2) Mark the unique elements in sortedArray[]:       
//          selectionArray[] = [ 1 0 1 0 0 1 0 1 1 0 ]
//
// (3) Scan selectionArray[] ("inclusive" all-prefix sums):  
//     scatterAddressArray[] = [ 1 1 2 2 2 3 3 4 5 5 ]
//
// (4) Scatter sortedArray[] into outputArray[] using scatterAddressArray[] - 1:
//             outputArray[] = [ 1 3 5 7 8 ]
//
// Note that the number of unique elements in the output array is the
// value of the last element in scatterAddressArray[].
//
// IMPORTANT: Step (1) to (4) must be computed on the GPU.
//
//===========================================================================
static void GPU_Unique( const int inputArray[], int numInputElems, 
                        int outputArray[], int *numUniqueElems )
{
    if ( numInputElems < 1 )
    {
        (*numUniqueElems) = 0;
        return;
    }

//---------------------------------------------------------------------------
// Allocate device memory and copy input array from host memory to
// device memory.
//---------------------------------------------------------------------------

    // Allocate device memory.
    int *d_sortedArray, *d_selectionArray, *d_scatterAddressArray, *d_outputArray;

    checkCudaErrors( hipMalloc( (void**) &d_sortedArray, numInputElems * sizeof(int) ) );
    checkCudaErrors( hipMalloc( (void**) &d_selectionArray, numInputElems * sizeof(int) ) );
    checkCudaErrors( hipMalloc( (void**) &d_scatterAddressArray, numInputElems * sizeof(int) ) );
    checkCudaErrors( hipMalloc( (void**) &d_outputArray, numInputElems * sizeof(int) ) );
    
    // Will contain the number of unique elements in the output array.
    int numSelectedElems = 0;

    // Copy host input array to device memory.
    checkCudaErrors( hipMemcpy( d_sortedArray, inputArray, numInputElems * sizeof(int), 
                     hipMemcpyHostToDevice ) );


//---------------------------------------------------------------------------
// Do Step (1) to (4).
//---------------------------------------------------------------------------

    //***********************************************
    //*********** WRITE YOUR CODE HERE **************
    //***********************************************

	// (0) Input array:  
	//              inputArray[] = [ 5 3 7 5 8 3 1 3 1 8 ]
	//
	
	// (1) Sort inputArray[]:   
	//             sortedArray[] = [ 1 1 3 3 3 5 5 7 8 8 ]
	GPU_SortIntegerArray(d_sortedArray, numInputElems);
	// (2) Mark the unique elements in sortedArray[]:       
	//          selectionArray[] = [ 1 0 1 0 0 1 0 1 1 0 ]
	Kernel_MarkUnique<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_sortedArray, d_selectionArray, numInputElems);
	
	// (3) Scan selectionArray[] ("inclusive" all-prefix sums):  
	//     scatterAddressArray[] = [ 1 1 2 2 2 3 3 4 5 5 ]
	GPU_AllPrefixSums(d_selectionArray, d_scatterAddressArray, numInputElems, &numSelectedElems);
	// (4) Scatter sortedArray[] into outputArray[] using scatterAddressArray[] - 1:
	//             outputArray[] = [ 1 3 5 7 8 ]
	Kernel_Scatter<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_sortedArray, d_selectionArray, d_scatterAddressArray, -1, d_outputArray, numInputElems);


//---------------------------------------------------------------------------
// Copy the final result from the device memory to the host memory.
//---------------------------------------------------------------------------

    checkCudaErrors( hipMemcpy( outputArray, d_outputArray, numSelectedElems * sizeof(int), 
                     hipMemcpyDeviceToHost ) );

    (*numUniqueElems) = numSelectedElems;


//---------------------------------------------------------------------------
// Clean up.
//---------------------------------------------------------------------------

    // Free device memory.
    checkCudaErrors( hipFree( d_sortedArray ) );
    checkCudaErrors( hipFree( d_selectionArray ) );
    checkCudaErrors( hipFree( d_scatterAddressArray ) );
    checkCudaErrors( hipFree( d_outputArray ) );
}




//===========================================================================
// Quicksort to sort the input integer array in-place in ascending order.
// To sort the entire input array, call Quicksort(array, 0, numElems-1).
//===========================================================================
#define SWAP(x, y, t)	((t)=(x),(x)=(y),(y)=(t))

static void Quicksort( int a[], int first, int last )
{
    int tmp; // Temporary variable for SWAP.

    if( first < last )
    {
        int pivot = a[first];
        int i = first - 1;
        int j = last + 1;

        while( true )
        {
            do { j--; } while ( a[j] > pivot );
            do { i++; } while ( a[i] < pivot );

            if( i < j ) 
                SWAP( a[i], a[j], tmp );
            else
                break;
        }

        Quicksort( a, first, j );
        Quicksort( a, j + 1, last );
    }
}

#undef SWAP



//===========================================================================
// CPU version.
//
// Given an input integer array, the function produces an output array
// which is a sorted version of the input array, but with duplicate 
// elements removed. The output array is sorted in non-decreasing order.
// The function also produces the number of unique elements in the
// output array in the parameter (*numUniqueElems).
//
// When this function is called, sufficient memory storage must have 
// already been allocated for the output array. The safest is to allocate
// as much memory as for the input array.
//===========================================================================
static void CPU_Unique( const int inputArray[], int numInputElems, 
                        int outputArray[], int *numUniqueElems )
{
    if ( numInputElems < 1 )
    {
        (*numUniqueElems) = 0;
        return;
    }

    int *sortedArray = (int *) malloc( numInputElems * sizeof(int) );
    memcpy( sortedArray, inputArray, numInputElems * sizeof(int) );

    Quicksort( sortedArray, 0, numInputElems - 1 );

    outputArray[0] = sortedArray[0];
    int uniqueCount = 1;

    for ( int i = 1; i < numInputElems; i++ )
        if ( sortedArray[i] != sortedArray[i-1] )
            outputArray[ uniqueCount++ ] = sortedArray[i];

    (*numUniqueElems) = uniqueCount;
}




//===========================================================================
// Generates a set of random integers, each has value from elemMin to
// elemMax, and put them in the array intArray[].
//===========================================================================
static void GenerateRandomIntegers( int intArray[], int numElems, int elemMin, int elemMax )
{
    for ( int i = 0; i < numElems; i++ )
    {
        int rand32 = rand() * (RAND_MAX + 1) + rand();
        intArray[i] = rand32 % (elemMax - elemMin + 1) + elemMin;
    }
}



//===========================================================================
// Return true iff all corresponding elements in the int 
// arrays A and B are equal.
//===========================================================================
static bool IntArrayEqual( const int A[], const int B[], int numElems )
{
    for ( int i = 0; i < numElems; i++ )
        if ( A[i] != B[i] ) return false;

    return true;
}



void WaitForEnterKeyBeforeExit( void )
{
    fflush( stdin );
    getchar();
}



//===========================================================================
// The main function
//===========================================================================
int main(int argc, char** argv)
{
    atexit( WaitForEnterKeyBeforeExit );

    // Set seed for rand().
    srand( 927 );

    // Use command-line specified CUDA device, otherwise use device with highest Gflops/s.
    int devID = findCudaDevice( argc, (const char **)argv );

    // Create a timer.
    StopWatchInterface *timer = 0;
    sdkCreateTimer( &timer );


//---------------------------------------------------------------------------
// Allocate host memory and generate test data.
//---------------------------------------------------------------------------

    // Allocate host memory for input integer array.
    int *inputArray = (int *) malloc( NUM_ELEMS * sizeof(int) );

    // Allocate host memory for result arrays.
    int *cpu_uniqueArray = (int *) malloc( NUM_ELEMS * sizeof(int) );
    int *gpu_uniqueArray = (int *) malloc( NUM_ELEMS * sizeof(int) );

    // Number of unique elements in input array computed by different methods.
    int cpu_numUniqueElems = 0;
    int gpu_numUniqueElems = 0;

    // Fill the input array with random integers.
    GenerateRandomIntegers( inputArray, NUM_ELEMS, ELEM_MIN, ELEM_MAX );


//---------------------------------------------------------------------------
// Print some program parameter values.
//---------------------------------------------------------------------------
    printf( "NUM_ELEMS  = %d\n", NUM_ELEMS );
    printf( "BLOCK_SIZE = %d\n", BLOCK_SIZE );
    printf( "ELEM_MIN   = %d\n", ELEM_MIN );
    printf( "ELEM_MAX   = %d\n", ELEM_MAX );
    printf( "\n\n" );


//---------------------------------------------------------------------------
// Perform computation on CPU.
//---------------------------------------------------------------------------
    printf( "CPU COMPUTATION:\n" );

    // Reset and start timer.
    sdkResetTimer( &timer );
    sdkStartTimer( &timer );

    // Compute on CPU.
    CPU_Unique( inputArray, NUM_ELEMS, cpu_uniqueArray, &cpu_numUniqueElems );

    // Stop timer.
    sdkStopTimer( &timer );
    printf( "Processing time = %.3f ms\n", sdkGetTimerValue( &timer ) );

    // Print some results.
    printf( "Number of unique elements = %d\n", cpu_numUniqueElems );
    printf( "\n\n" );


//---------------------------------------------------------------------------
// Perform computation on GPU.
//---------------------------------------------------------------------------
    printf( "GPU COMPUTATION:\n" );

    // Reset and start timer.
    sdkResetTimer( &timer );
    sdkStartTimer( &timer );

    // Compute on GPU.
    GPU_Unique( inputArray, NUM_ELEMS, gpu_uniqueArray, &gpu_numUniqueElems );

    // Stop timer.
    sdkStopTimer( &timer );
    printf( "Processing time = %.3f ms\n", sdkGetTimerValue( &timer ) );

    // Print some results.
    printf( "Number of unique elements = %d\n", gpu_numUniqueElems );
    printf( "\n" );

    // Check result with reference result computed by CPU.
    bool equal = ( gpu_numUniqueElems == cpu_numUniqueElems ) && 
                  IntArrayEqual( cpu_uniqueArray, gpu_uniqueArray, cpu_numUniqueElems );
    printf( "Verify GPU result... %s\n", (equal)? "PASS" : "FAIL" );
    printf( "\n\n" );


//---------------------------------------------------------------------------
// Clean up.
//---------------------------------------------------------------------------
    // Destroy the timer.
    sdkDeleteTimer( &timer );

    // Free up memory.
    free( inputArray );
    free( cpu_uniqueArray );
    free( gpu_uniqueArray );

    hipDeviceReset();
}
